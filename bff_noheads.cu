// Copyright 2024 Google LLC
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cstddef>
#include <cstdint>

#include "common_language.h"

namespace {

__host__ __device__ bool isin(const char *chars, char c) {
  for (; *chars; chars++) {
    if (c == *chars) return true;
  }
  return false;
}
// Core data structure representing the BFF language interpreter
struct Bff {
  static const char *name() { return "bff_noheads"; }

  // Initialize color coding for different characters in visualization
  static void InitByteColors(
      std::array<std::array<uint8_t, 3>, 256> &byte_colors) {
    
    // Set default gray colors for all characters
    for (size_t i = 0; i < 256; i++) {
      const uint8_t v = 192 + i / 4;
      byte_colors[i] = {v, v, v};
    }
    // Special colors for different command types:
    byte_colors[0] = {255, 0, 0};
    byte_colors['['] = byte_colors[']'] = {0, 192, 0};
    byte_colors['+'] = byte_colors['-'] = {200, 0, 200};
    byte_colors['.'] = byte_colors[','] = {200, 0, 200};
    byte_colors['<'] = byte_colors['>'] = {0, 128, 220};
    byte_colors['{'] = byte_colors['}'] = {0, 128, 220};
  }
  
  // Parse input string, handling special null character representation
  static std::string Parse(std::string bff) {
    std::string ret;
    for (size_t i = 0; i < bff.size();) {
      if (bff.substr(i, 3) == "␀") {
        // Handle special null character encoding
        ret.push_back(0);
        i += 3;
      } else {
        ret.push_back(bff[i]);
        i++;
      }
    }
    return ret;
  }

  static __device__ __host__ const char *MapChar(char c, char *chmem) {
    constexpr const char *kPrintable =
        " !\"#$%&'()*+,-./"
        "0123456789:;<=>?@ABCDEFGHIJKLMNOPQRSTUVWXYZ[\\]^_`"
        "abcdefghijklmnopqrstuvwxyz{|}~";
    chmem[1] = 0;
    chmem[0] = c;
    if (chmem[0] == 0) {
      return "␀";
    }
    if (!isin(kPrintable, chmem[0])) {
      chmem[0] = ' ';
    }
    return chmem;
  }

  static __device__ __host__ void PrintProgramInternal(
      size_t head0_pos, size_t head1_pos, size_t pc_pos, const uint8_t *mem,
      size_t len, const uint8_t *mem2, size_t len2) {
    auto print_char = [&](char c, size_t i) {
      char chmem[32] = {};
      const char *cc = MapChar(mem[i], chmem);
      bool is_command = isin("<>{}+-.,[]", *cc);
      if (i == head0_pos) {
        printf("\x1b[44;1m");
      }
      if (i == head1_pos) {
        printf("\x1b[41;1m");
      }
      if (i == pc_pos) {
        printf("\x1b[42;1m");
      }
      if (is_command) {
        printf("\x1b[37;1m");
      }
      printf("%s", cc);
      if (is_command || i == head0_pos || i == head1_pos || i == pc_pos) {
        printf("\x1b[;m");
      }
    };
    for (size_t i = 0; i < len; i++) {
      char c = mem[i];
      print_char(c, i);
    }
    if (mem2) {
      printf("   ");
      for (size_t i = len; i < len + len2; i++) {
        char c = mem2[i - len];
        print_char(c, i);
      }
    }
    printf("\n");
  }

  static void PrintProgram(size_t pc_pos, const uint8_t *mem, size_t len,
                           const uint8_t *mem2, size_t len2) {
    size_t head0_pos = 2 * kSingleTapeSize;
    size_t head1_pos = 2 * kSingleTapeSize;
    PrintProgramInternal(head0_pos, head1_pos, pc_pos, mem, len, mem2, len2);
  }

  // Main interpreter function
  static __device__ size_t Evaluate(uint8_t *tape, size_t stepcount,
                                    bool debug) {
    size_t nskip = 0;

    int pos = 0; // instruction pointer
    int head0_pos = 0; // primary tape head pointer
    int head1_pos = 0; // second tape head pointer

    size_t i = 0;
    
    // Main execution loop
    for (; i < stepcount; i++) {
      // Wrap head positions within tape bounds
      head0_pos = head0_pos & (2 * kSingleTapeSize - 1); 
      head1_pos = head1_pos & (2 * kSingleTapeSize - 1);
      if (debug) {
        PrintProgramInternal(head0_pos, head1_pos, pos, tape,
                             2 * kSingleTapeSize, nullptr, 0);
      }
      char cmd = tape[pos];
      switch (cmd) {
        case '<':
          head0_pos--;
          break;
        case '>':
          head0_pos++;
          break;
        case '{':
          head1_pos--;
          break;
        case '}':
          head1_pos++;
          break;
        case '+':
          tape[head0_pos]++;
          break;
        case '-':
          tape[head0_pos]--;
          break;
        case '.':
          tape[head1_pos] = tape[head0_pos];
          break;
        case ',':
          tape[head0_pos] = tape[head1_pos];
          break;
        case '[':
          if (!tape[head0_pos]) {
            size_t scanclosed = 1;
            pos++;
            for (; pos < (2 * kSingleTapeSize) && scanclosed > 0; pos++) {
              if (tape[pos] == ']') scanclosed--;
              if (tape[pos] == '[') scanclosed++;
            }
            pos--;
            if (scanclosed != 0) {
              pos = 2 * kSingleTapeSize;
            }
          }
          break;
        case ']':
          if (tape[head0_pos]) {
            size_t scanopen = 1;
            pos--;
            for (; pos >= 0 && scanopen > 0; pos--) {
              if (tape[pos] == ']') scanopen++;
              if (tape[pos] == '[') scanopen--;
            }
            pos++;
            if (scanopen != 0) {
              pos = -1;
            }
          }
          break;
        default:
          nskip++;
      }
      if (pos < 0) {
        i++;
        break;
      }
      pos++;
      if (pos >= 2 * kSingleTapeSize) {
        i++;
        break;
      }
    }

    return i - nskip;
  }
};

REGISTER(Bff);
}  // namespace
